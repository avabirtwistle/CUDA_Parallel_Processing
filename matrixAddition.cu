#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addArrays(int *a, int *b, int *c, int n){
    //define a unique identifier for each thread 
    //each thread will compute one element of the array
      int index = threadIdx.x + blockIdx.x * blockDim.x; //ID of thread within a block, ID of thread within the grid, number of blocks in the grid
  
  // check if the thread index is within the bounds of the array
  if(index<n){
          c[index] = a[index] + b[index]; // add a and b element wise and save this to the result array
      }
}

int main(){
  const int N = 512; // number of elements in the array
  int h_a[N], h_b[N], h_c[N]; // initiallize the ararys 

  for(int i = 0; i<N; i++){ // populate the arrays
    h_a[i] = i;
    h_b[i] = i*2;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, sizeof(int)*N); //allocate memory on the device for array a
  hipMalloc(&d_b, sizeof(int)*N); //allocate memory on the device for array b
  hipMalloc(&d_c, sizeof(int)*N); //allocate memory on the device for array c (result array)

  //Transfer data from CPU to GPU
  hipMemcpy(d_a, h_a, sizeof(int)*N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, sizeof(int)*N, hipMemcpyHostToDevice);

  //Launch the kernel on the GPU
  addArrays<<<N/256, 256>>>(d_a, d_b, d_c, N);

  hipMemcpy(h_c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost); // copy the results stored in d_c on the GPU back to h_c on the CPU

  //Print the results (first 10)
  for(int i = 0; i<10; i++){
    std::cout << h_a[i] << "+" <<h_b[i]<< "=" << h_c[i]<<std::endl;
  }

  //Free the memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}
